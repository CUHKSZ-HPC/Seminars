#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if (error != hipSuccess) \
    { \
        printf("Error: %s:%d, ", __FILE__, __LINE__); \
        printf("code:%d, reason: %s\n", error, hipGetErrorString(error)); \
        exit(1); \
    } \
}

void checkResult(float *hostRef, float *gpuRef, const int N) {
    double epsilon = 1.0E-8;
    bool match = 1;
    for (int i=0; i<N; i++) {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0;
            printf("Arrays do not match!\n");
            printf("host %5.2f gpu %5.2f at current %d\n",hostRef[i],gpuRef[i],i);
            break;
        }
    }
    if (match) printf("Arrays match.\n\n");
}

void initialData(float *ip,int size) {
    // generate different seed for random number
    time_t t;
    srand((unsigned) time(&t));
    for (int i=0; i<size; i++) {
        ip[i] = (float)( rand() & 0xFF )/10.0f;
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N) {
    for (int idx=0; idx<N; idx++)
        C[idx] = A[idx] + B[idx];
}

__global__ void sumArraysZeroCopy(float *A, float *B, float *C, const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) C[i] = A[i] + B[i];
}

int main(int argc, char **argv) {
    // part 0: set up device and array
    // set up device
    int dev = 0;
    hipSetDevice(dev);
    
    // get device properties
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    
    // check if support mapped memory
    if (!deviceProp.canMapHostMemory) {
        printf("Device %d does not support mapping CPU host memory!\n", dev);
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }
    printf("Using Device %d: %s ", dev, deviceProp.name);
    
    // set up date size of vectors
    int ipower = 10;
    if (argc>1) ipower = atoi(argv[1]);
    int nElem = 1<<ipower;
    size_t nBytes = nElem * sizeof(float);
    if (ipower < 18) {
        printf("Vector size %d power %d nbytes %3.0f KB\n", nElem,\
        ipower,(float)nBytes/(1024.0f));
    } else {
        printf("Vector size %d power %d nbytes %3.0f MB\n", nElem,\
        ipower,(float)nBytes/(1024.0f*1024.0f));
    }
    
    // part 1: using device memory
    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);
    
    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);
    
    // add vector at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    
    // malloc device global memory
    float *d_A, *d_B, *d_C;
    hipMalloc((float**)&d_A, nBytes);
    hipMalloc((float**)&d_B, nBytes);
    hipMalloc((float**)&d_C, nBytes);
    
    // transfer data from host to device
    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
    
    // set up execution configuration
    int iLen = 512;
    dim3 block (iLen);
    dim3 grid ((nElem+block.x-1)/block.x);
    
    // invoke kernel at host side
    sumArrays <<<grid, block>>>(d_A, d_B, d_C, nElem);
    
    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
    
    // check device results
    checkResult(hostRef, gpuRef, nElem);
    
    // free device global memory
    hipFree(d_A);
    hipFree(d_B);
    free(h_A);
    free(h_B);
    
    // part 2: using zerocopy memory for array A and B
    // allocate zerocpy memory
    unsigned int flags = hipHostMallocMapped;
    hipHostAlloc((void **)&h_A, nBytes, flags);
    hipHostAlloc((void **)&h_B, nBytes, flags);
    
    // initialize data at host side
    initialData(h_A, nElem);
    initialData(h_B, nElem);
    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);
    
    // pass the pointer to device
    hipHostGetDevicePointer((void **)&d_A, (void *)h_A, 0);
    hipHostGetDevicePointer((void **)&d_B, (void *)h_B, 0);
    
    // add at host side for result checks
    sumArraysOnHost(h_A, h_B, hostRef, nElem);
    
    // execute kernel with zero copy memory
    sumArraysZeroCopy <<<grid, block>>>(d_A, d_B, d_C, nElem);
    
    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
    
    // check device results
    checkResult(hostRef, gpuRef, nElem);
    
    // free memory
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    free(hostRef);
    free(gpuRef);
    
    // reset device
    hipDeviceReset();
    return EXIT_SUCCESS;
}